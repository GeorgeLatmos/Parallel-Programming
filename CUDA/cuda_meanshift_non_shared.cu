
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

const int N = 600;
const int D = 2;
const int blocksize = 4;
const int gridsize = 4;

struct timeval startwtime, endwtime;
double seq_time;

double **alloc_2d_init(int rows, int cols);
double **init();

__device__
void findNext(double *a,double *b, double *c,int index){
	
	double numinator;
    double denuminator;
    double distance;
    double sigma = 1.0;
    double c1;
    int j,k,d;
	
    for(j=0; j<D; j++){
       	numinator = 0.0;
       	denuminator = 0.0;
       	for(k=0; k<N; k++){
           distance = 0.0;
    	   for(d=0; d<D; d++){
    		  distance += (b[index*D+d]-a[k*D+d])*(b[index*D+d]-a[k*D+d]);
    	   }
         if(distance<sigma*sigma){
           c1 = exp(-distance/(2*sigma*sigma));
           denuminator += c1;
           numinator += c1*a[k*D+j];
         }
    	   
    	}
    	c[index*D+j] = numinator/denuminator;
    }
}

__global__
void meanshiftNonShared(double *a, double *b, double *c){

	int index = blockIdx.x*blockDim.x + threadIdx.x;
	int i,j;

  double abs_m;

  double epsilon = 0.0001;

  for(i=index*N/(blockDim.x*blockDim.x); i<(index+1)*N/(blockDim.x*blockDim.x); i++){
        
    abs_m = 1.0;

    while(abs_m > epsilon){

    	findNext(a,b,c,i);

    	abs_m = 0.0;
    	for(j=0; j<D; j++){
    	   abs_m += (c[i*D+j]-b[i*D+j])*(c[i*D+j]-b[i*D+j]);
      }
      for(j=0; j<D; j++){
          b[i*D+j] = c[i*D+j];
      }

    }
  }
}

int main(){

	double **points;
	double **new_points;

	double *lin_points, *lin_prev, *lin_next;

	lin_points = (double *)malloc(N*D*sizeof(double));
	lin_prev = (double *)malloc(N*D*sizeof(double));
	lin_next = (double *)malloc(N*D*sizeof(double));

	new_points = alloc_2d_init(N,D);

	points = init();

	int i,j,index;

	for(i=0; i<N; i++){
		for(j=0; j<D; j++){
			index = j+i*D;
			lin_points[index] = points[i][j];
			lin_prev[index] = points[i][j];
		}
	}

	double *ad, *bd, *cd;
	const int size = N*D*sizeof(double);

	hipMalloc((void **)&ad,size);
	hipMalloc((void **)&bd,size);
	hipMalloc((void **)&cd,size);

	hipMemcpy(ad,lin_points,size,hipMemcpyHostToDevice);
	hipMemcpy(bd,lin_prev,size,hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);
    
    gettimeofday (&startwtime, NULL);

	meanshiftNonShared<<<dimGrid,dimBlock>>>(ad,bd,cd);

	hipDeviceSynchronize();

	gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
        + endwtime.tv_sec - startwtime.tv_sec);

  printf("Meanshift Non Shared wall clock time = %f\n\n", seq_time);

  hipMemcpy(lin_next,cd,size,hipMemcpyDeviceToHost);
  
	for(i=0; i<N; i++){
		for(j=0; j<D; j++){
			new_points[i][j] = lin_next[j+i*D];
		}
	}

  hipFree(ad); 
  hipFree(bd);
  hipFree(cd);

	return EXIT_SUCCESS;
}

double **init(){

   FILE *fp;
   double *buffer;
   double **ret_buf;
   int size,number_of_elements;
   int i,j,counter;

   fp = fopen("data.bin","rb");

   fseek(fp, 0, SEEK_END);

   size = ftell(fp); //total size in bytes

   number_of_elements = size/8; //total number of elements

   buffer = (double *)malloc(number_of_elements*sizeof(double));

   ret_buf = alloc_2d_init(N,D);

   fseek(fp, 0, SEEK_SET);

   fread(buffer,sizeof(buffer),number_of_elements,fp);

   fclose(fp);

   counter = 0;
   for(j=0; j<D; j++){
    for(i=0; i<N; i++){
      ret_buf[i][j] = buffer[counter];
      counter++;
    }
   }
   return ret_buf;
}

double **alloc_2d_init(int rows, int cols){
    double *data = (double *)malloc(rows*cols*sizeof(double));
    double **array= (double **)malloc(rows*sizeof(double*));
    int i;
    for (i=0; i<rows; i++)
        array[i] = &(data[cols*i]);

    return array;
}